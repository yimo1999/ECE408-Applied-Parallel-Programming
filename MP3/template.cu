#include "hip/hip_runtime.h"

#include <wb.h>

#define Tile_width 8
#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)

// Compute C = A * B
__global__ void matrixMultiply(float *A, float *B, float *C, int numARows,
                               int numAColumns, int numBRows,
                               int numBColumns, int numCRows,
                               int numCColumns) {
  //@@ Insert code to implement matrix multiplication here

  __shared__ float subTileM[Tile_width][Tile_width];
  __shared__ float subTileN[Tile_width][Tile_width];

  int bx = blockIdx.x;
  int by = blockIdx.y;
  int tx = threadIdx.x;
  int ty = threadIdx.y;

  int row = by * Tile_width + ty;
  int col = bx * Tile_width + tx;

  float Pvalue = 0;

  for(int q = 0; q < (ceil((float)numAColumns/Tile_width)); q++){
    if(row < numARows && (q * Tile_width + tx) < numAColumns){
      subTileM[ty][tx] = A[row * numAColumns + (q * Tile_width + tx)];
    }else{
      subTileM[ty][tx] = 0;
    }

    if(col < numBColumns && (q * Tile_width + ty < numBRows)){
      subTileN[ty][tx] = B[col + (q * Tile_width + ty) * numBColumns];
    }else{
      subTileN[ty][tx] = 0;
    }

    __syncthreads();

    for(int i = 0; i < Tile_width; i++){
      Pvalue += subTileM[ty][i] * subTileN[i][tx];
    }

    __syncthreads();

    if(row < numARows && col < numBColumns){
      C[row * numBColumns + col] = Pvalue;
    }

    }
  }


int main(int argc, char **argv) {
  wbArg_t args;
  float *hostA; // The A matrix
  float *hostB; // The B matrix
  float *hostC; // The output C matrix
  float *deviceA;
  float *deviceB;
  float *deviceC;
  int numARows;    // number of rows in the matrix A
  int numAColumns; // number of columns in the matrix A
  int numBRows;    // number of rows in the matrix B
  int numBColumns; // number of columns in the matrix B
  int numCRows;    // number of rows in the matrix C (you have to set this)
  int numCColumns; // number of columns in the matrix C (you have to set
                   // this)

  args = wbArg_read(argc, argv);

  wbTime_start(Generic, "Importing data and creating memory on host");
  hostA = (float *)wbImport(wbArg_getInputFile(args, 0), &numARows,
                            &numAColumns);
  hostB = (float *)wbImport(wbArg_getInputFile(args, 1), &numBRows,
                            &numBColumns);
  //@@ Set numCRows and numCColumns
  // numCRows = 0;
  // numCColumns = 0;

  numCRows = numARows;
  numCColumns = numBColumns;
  //@@ Allocate the hostC matrix
  hostC = (float *)malloc(numCRows * numCColumns * sizeof(float));

  wbTime_stop(Generic, "Importing data and creating memory on host");

  wbLog(TRACE, "The dimensions of A are ", numARows, " x ", numAColumns);
  wbLog(TRACE, "The dimensions of B are ", numBRows, " x ", numBColumns);

  wbTime_start(GPU, "Allocating GPU memory.");
  //@@ Allocate GPU memory here
  hipMalloc((void **)&deviceA, numARows * numAColumns * sizeof(float));
  hipMalloc((void **)&deviceB, numBRows * numBColumns * sizeof(float));
  hipMalloc((void **)&deviceC, numCRows * numCColumns * sizeof(float));

  wbTime_stop(GPU, "Allocating GPU memory.");

  wbTime_start(GPU, "Copying input memory to the GPU.");
  //@@ Copy memory to the GPU here
  hipMemcpy(deviceA, hostA, numARows * numAColumns * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(deviceB, hostB, numBRows * numBColumns * sizeof(float), hipMemcpyHostToDevice);

  wbTime_stop(GPU, "Copying input memory to the GPU.");

  //@@ Initialize the grid and block dimensions here
  int Block_width = 8;
  dim3 DimBlock(Block_width, Block_width, 1);
  dim3 DimGrid(ceil((numCColumns * 1.0) / Block_width), ceil((numCRows * 1.0) / Block_width), 1);
  wbTime_start(Compute, "Performing CUDA computation");
  //@@ Launch the GPU Kernel here
  matrixMultiply<<<DimGrid, DimBlock>>>(deviceA, deviceB, deviceC, numARows, numAColumns, numBRows, numBColumns, numCRows, numCColumns);


  hipDeviceSynchronize();
  wbTime_stop(Compute, "Performing CUDA computation");

  wbTime_start(Copy, "Copying output memory to the CPU");
  //@@ Copy the GPU memory back to the CPU here
  hipMemcpy(hostC, deviceC, numCRows * numCColumns * sizeof(float), hipMemcpyDeviceToHost);


  wbTime_stop(Copy, "Copying output memory to the CPU");

  wbTime_start(GPU, "Freeing GPU Memory");
  //@@ Free the GPU memory here
  hipFree(deviceA);
  hipFree(deviceB);
  hipFree(deviceC);

  wbTime_stop(GPU, "Freeing GPU Memory");

  wbSolution(args, hostC, numCRows, numCColumns);

  free(hostA);
  free(hostB);
  free(hostC);

  return 0;
}
